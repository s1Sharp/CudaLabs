#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <iostream>
#include <thread>
#include <functional>

void print_info(FILE *);

__global__ void add(int* a, int* b, int* c) 
{
    *c = *a + *b;
}

FILE* file = fopen("output.txt", "w");

int main() 
{
    

    int a, b, c;
    // host copies of variables a, b & c
    int* d_a, * d_b, * d_c;
    // device copies of variables a, b & c
    int size = sizeof(int);
    // Allocate space for device copies of a, b, c
    hipMalloc((void**)&d_a, size);
    hipMalloc((void**)&d_b, size);
    hipMalloc((void**)&d_c, size);
    // Setup input values  
    c = int();
    a = 7777u;
    b = 333u;
    // Copy inputs to device
    hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);
    // Launch add() kernel on GPU
    add << <1, 1 >> > (d_a, d_b, d_c);
    // Copy result back to host
    hipError_t err = hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);
    if (err != hipSuccess) 
    {
        fprintf(file,"CUDA error copying to Host: %s\n", hipGetErrorString(err));
    }
    fprintf(file,"result sum of %d and %d is %d\n",a , b, c);

   
    // Cleanup
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    return 0;
}

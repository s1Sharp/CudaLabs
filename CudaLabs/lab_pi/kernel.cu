#include "hip/hip_runtime.h"
﻿#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""
#include "hiprand/hiprand_kernel.h"

#include <memory>
#include <ctime>
#include <stdio.h>
#include <chrono>
#include <algorithm>

#define ACCURACY 12828879
#define NUM_OF_ITER 10


using defer = std::shared_ptr<void>;

//cuRandom API prototype 
//                   __device__ float hiprand_uniform(hiprandState_t* state) ->
//single normally distributed float with mean 0.0 and standard deviation 1.0

__global__ void Pi(int* count, hiprandState_t* globalState,unsigned long long seed,int accuracy)
{
	int indx = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;

	hiprand_init(seed, indx, 0, &globalState[indx]);

	hiprandState_t localState = globalState[indx];
	while (indx < accuracy) 
	{
		float temp_x = hiprand_uniform(&localState);
		float temp_y = hiprand_uniform(&localState);
		float z = temp_x * temp_x + temp_y * temp_y;
		if (z < 1.0f) {
			atomicAdd(count, 1);
		}
		indx += stride;
	}
}

int main()
{
	int count;
	int* dev_count;
	float res = 0.0f;
	float* sort_res = new float[NUM_OF_ITER];

	unsigned long long seed = time(NULL);

	hipDeviceProp_t  prop;
	hipGetDeviceProperties(&prop, 0);
	
	//max Speed with blocks * 2
	int blocks = prop.multiProcessorCount *  2;
	int threads = prop.maxThreadsPerBlock;
	int total = (ACCURACY - (ACCURACY % threads)); //last indx of __global__ func
	printf("kernel start with %u blocks and %u threads, total %u\n", blocks, threads,total);

	//for random eq
	hiprandState_t* devState;
	hipMalloc((void**)&devState, total * sizeof(hiprandState_t));

	//allocate memory on GPU
	hipMalloc((void**)&dev_count, sizeof(int));

	// starting the timer here so that we include the cost of
	// all of the operations on the GPU.
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);


	//use un_ptr, that don`t forget free memory
	defer _(nullptr, [dev_count, devState, start, stop,sort_res ](...)
		{ hipFree(dev_count); hipFree(devState); hipEventDestroy(start); hipEventDestroy(stop); delete[] sort_res; printf("free"); });


	for (int iter = 0; iter < NUM_OF_ITER; iter++)
	{
		//kernel
		Pi << < blocks, threads >> > (dev_count, devState, seed, total);

		//copy result to HOST
		hipMemcpy(&count, dev_count, sizeof(int), hipMemcpyDeviceToHost);

		//upd seed
		seed = std::chrono::duration_cast<std::chrono::milliseconds>
															(std::chrono::system_clock::now().time_since_epoch()).count() + iter;
		hipMemset(dev_count, 0, sizeof(int));

		float tempres = count * 4.0f / total;

		printf("res of %u iter PI: %f \n", iter, tempres);
		sort_res[iter] = tempres;
		res += tempres;
	}

	std::sort(sort_res, sort_res + 10);
	printf("______sorted result_____ \n");
	for (size_t i = 0; i < 10; i++)
	{
		printf("\tPI: %f \n",*(sort_res + i));
	}

	//print elapsed time
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float   elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);
	printf("Time to generate:  %3.1f ms in mean foreach iter, total time: %3.1f ms \n", elapsedTime/NUM_OF_ITER, elapsedTime);
	
	//result
	printf("result = %f\n", res / NUM_OF_ITER);

	return 0;
}

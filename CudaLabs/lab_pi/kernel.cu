#include "hip/hip_runtime.h"
﻿#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""
#include "hiprand/hiprand_kernel.h"

#include <memory>
#include <ctime>
#include <stdio.h>


#define ACCURACY 12828849
#define NUM_OF_ITER 10

using defer = std::shared_ptr<void>;

//cuRandom API prototype 
//                   __device__ float hiprand_uniform(hiprandState_t* state) ->
//single normally distributed float with mean 0.0 and standard deviation 1.0

__global__ void Pi(int* count, hiprandState_t* globalState,unsigned int seed,int accuracy)
{
	int indx = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;

	hiprand_init(seed, indx, 0, &globalState[indx]);
	//printf("idx = %u \n", indx);

	hiprandState_t localState = globalState[indx];
	while (indx < accuracy) {
		

		float temp_x = hiprand_uniform(&localState);
		float temp_y = hiprand_uniform(&localState);

		float z = temp_x * temp_x + temp_y * temp_y; 
		//printf("x = %f y = %f z = %f\n", temp_x, temp_y,z); 
		if (z < 1.0f) {
			atomicAdd(count, 1);
		}
		//printf("indx = %u , count = %u\n", indx, *count);
		indx +=stride;
	}
	//printf("max = %u \n", indx-stride);
}

int main()
{
	int count;
	int* dev_count;
	float res = 0.0f;

	unsigned int seed = time(NULL);

	hipDeviceProp_t  prop;
	hipGetDeviceProperties(&prop, 0);
	
	//max Speed with blocks * 2
	int blocks = prop.multiProcessorCount * 2; 
	int threads = prop.maxThreadsPerBlock;
	int total = (ACCURACY - (ACCURACY % threads)); //last indx of __global__ func
	printf("kernel start with %u blocks and %u threads, total %u\n", blocks, threads,total);



	//for random eq
	hiprandState_t* devState;
	hipMalloc((void**)&devState, total * sizeof(hiprandState_t));



	//allocate memory on GPU
	hipMalloc((void**)&dev_count, sizeof(int));

	//use un_ptr, that don`t forget free memory
	defer _(nullptr, [dev_count, devState](...)
		{ hipFree(dev_count); hipFree(devState);  printf("free"); });

	// starting the timer here so that we include the cost of
	// all of the operations on the GPU.
	hipEvent_t     start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	
	for (int iter = 0; iter < NUM_OF_ITER; iter++)
	{

		//kernel
		Pi << < blocks, threads >> > (dev_count, devState, seed, total);

		//copy result to HOST
		hipMemcpy(&count, dev_count, sizeof(int), hipMemcpyDeviceToHost);

		//upd seed
		seed = time(NULL);
		hipMemset(dev_count, 0, sizeof(int));

		float tempres = count * 4.0f / total;
		res += tempres;
	}

	//print elapsed time
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float   elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);
	printf("Time to generate:  %3.1f ms in mean foreach iter, total time: %3.1f ms \n", elapsedTime/NUM_OF_ITER, elapsedTime);


	printf("result = %f\n", res / NUM_OF_ITER);

	return 0;
}

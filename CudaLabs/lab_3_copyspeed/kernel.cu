﻿
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#include <iostream>

// 1gb = 1073741824 bytes; float = 4 bytes; => size = 268435456 size_t 
const size_t size = 1024 * 1024 * 256;

const size_t count = 25;

int main()
{
    // initialize clocks 
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);


    float* dev_tmp;
    float* tmp = (float*)malloc( size* sizeof(float));
    hipMalloc((void**)&dev_tmp,size * sizeof(float));


    hipEventRecord(start, 0);

    for (size_t iter = 0; iter < count; iter++) {
        hipMemcpy(dev_tmp, tmp, size * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(tmp, dev_tmp, size * sizeof(float), hipMemcpyDeviceToHost);
    }
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    float   elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);

    float speed = 2.f * 1000 * 25 / elapsedTime; // GB per sec
    std::cout <<  speed << " GB/sec";


    hipFree(dev_tmp);
    free(tmp);

    return 0;
}


﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include "matrix.h"

template <typename T>
void kernel(mymatrix::MATRIX<T> m1, mymatrix::MATRIX<T> m2)
{
    m1 += m2;
};

int main()
{
    using namespace mymatrix;
    mymatrix::test();


    return 0;
}


﻿#include <hip/hip_runtime.h>


#include "hiprand/hiprand_kernel.h"

#include <memory>
#include <ctime>
#include <stdio.h>
#include <chrono>
#include <algorithm>


using defer = std::shared_ptr<void>;

#define WIDTH 32
#define HEIGHT 32
#define hBLOCKS 6
#define wBLOCKS 6

//return resurl of this expression (x*x + y*y <= 1)
__device__ bool inCircle(hiprandState_t* state)
{
    float x = hiprand_uniform(state);
    float y = hiprand_uniform(state);
    return x * x + y * y <= 1.0f;
}


__global__ void CalculatePointsIntheCircle(int* result, int randseed)
{
    hiprandState_t state;
    (threadIdx.x + blockDim.x * blockIdx.x)* threadIdx.y + blockDim.y * blockIdx.y;
    unsigned long long seed = (threadIdx.x + blockDim.x * blockIdx.x) + (threadIdx.y + blockDim.y * blockIdx.y) * (randseed % 1000);

    //init curand
    hiprand_init(seed, 0, 0, &state);

    if (inCircle(&state))
    {
        atomicAdd(result, 1);
    }
}

int main()
{
    int count = 0;
    int* dev_count;

    hipMalloc((void**)&dev_count, sizeof(int));
    
    // starting the timer here so that we include the cost of
    // all of the operations on the GPU.
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    //use un_ptr, that don`t forget free memory
    defer _(nullptr, [&](...)
        { hipFree(dev_count);  hipEventDestroy(start); hipEventDestroy(stop);  printf("free"); });

    dim3 blocks(hBLOCKS, wBLOCKS, 1);
    dim3 threads(HEIGHT, WIDTH, 1);

    int randseed = std::chrono::duration_cast<std::chrono::milliseconds>
        (std::chrono::system_clock::now().time_since_epoch()).count();

    CalculatePointsIntheCircle <<<blocks, threads >>> (dev_count, randseed);

    hipMemcpy(&count, dev_count, sizeof(int), hipMemcpyDeviceToHost);
    // result pi
    float pi = (4.0f * static_cast<float>(count)) / static_cast<float>(HEIGHT * WIDTH * hBLOCKS * wBLOCKS);
    printf("Result pi %f \n", pi);

    //print elapsed time
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float   elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf( "Elapsed time %3.1f ms\n", elapsedTime );

    return 0;
}
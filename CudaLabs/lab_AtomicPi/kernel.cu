﻿#include <hip/hip_runtime.h>


#include "hiprand/hiprand_kernel.h"

#include <memory>
#include <ctime>
#include <stdio.h>
#include <stdlib.h>
#include <chrono>
#include <algorithm>
#include <numeric>   


using defer = std::shared_ptr<void>;

#define HEIGHT 32
#define WIDTH 32
#define hBLOCKS 6
#define wBLOCKS 6

//return resurl of this expression (x*x + y*y <= 1)
__device__ bool inCircle(hiprandState_t* state)
{
    float x = hiprand_uniform(state);
    float y = hiprand_uniform(state);
    return x * x + y * y <= 1.0f;
}


__global__ void CalculatePointsIntheCircle(int* result, int randseed)
{
    hiprandState_t state;
    unsigned long long seed = (threadIdx.x + blockDim.x * blockIdx.x) + (threadIdx.y + blockDim.y * blockIdx.y) * (randseed % 1000);

    //init curand
    hiprand_init(seed, 0, 0, &state);

    if (inCircle(&state))
    {
        atomicAdd(&result[threadIdx.x * HEIGHT + threadIdx.y], 1);
    }
    return;
}

int main()
{
    const size_t size = WIDTH * HEIGHT;
    int count [size];
    memset(&count, 0, size * sizeof(int));
    int* dev_count;

    hipMalloc((void**)&dev_count, size * sizeof(int));
    
    // starting the timer here so that we include the cost of
    // all of the operations on the GPU.
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    //use un_ptr, that don`t forget free memory
    defer _(nullptr, [&](...)
        { hipFree(dev_count);  hipEventDestroy(start); hipEventDestroy(stop);  printf("free"); });

    dim3 blocks(hBLOCKS, wBLOCKS, 1);
    dim3 threads(HEIGHT, WIDTH, 1);

    int randseed = std::chrono::duration_cast<std::chrono::milliseconds>
        (std::chrono::system_clock::now().time_since_epoch()).count();

    CalculatePointsIntheCircle <<<blocks, threads >>> (dev_count, randseed);

    hipMemcpy(&count, dev_count, size * sizeof(int), hipMemcpyDeviceToHost);
    // result pi

    int ans = 0;
    ans = std::accumulate(&count[0], &count[size - 1], ans);

    float fullsize = static_cast<float>(HEIGHT * WIDTH * hBLOCKS * wBLOCKS);
    float pi = (4.0f * static_cast<float>(ans));
    pi /=  fullsize;
    printf("Result pi %f \n", pi);

    //print elapsed time
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float   elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf( "Elapsed time %3.1f ms\n", elapsedTime );

    return 0;
}
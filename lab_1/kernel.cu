#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

void print_info();

__global__ void add(int* a, int* b, int* c) 
{
    *c = *a + *b;
}

int main() 
{
    int a, b, c;
    // host copies of variables a, b & c
    int* d_a, * d_b, * d_c;
    // device copies of variables a, b & c
    int size = sizeof(int);
    // Allocate space for device copies of a, b, c
    hipMalloc((void**)&d_a, size);
    hipMalloc((void**)&d_b, size);
    hipMalloc((void**)&d_c, size);
    // Setup input values  
    c = int();
    a = 777u;
    b = 333u;
    // Copy inputs to device
    hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);
    // Launch add() kernel on GPU
    add << <1, 1 >> > (d_a, d_b, d_c);
    // Copy result back to host
    hipError_t err = hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);
    if (err != hipSuccess) 
    {
        printf("CUDA error copying to Host: %s\n", hipGetErrorString(err));
    }
    printf("result sum of %d and %d is %d\n",a , b, c);
    print_info();
    // Cleanup
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    return 0;
}

void print_info()
{
    int            deviceCount;
    hipDeviceProp_t devProp;

    hipGetDeviceCount(&deviceCount);

    printf("Found %d devices\n", deviceCount);

    for (int device = 0; device < deviceCount; device++)
    {
        hipGetDeviceProperties(&devProp, device);

        printf("Device %d\n", device);
        printf("Compute capability     : %d.%d\n", devProp.major, devProp.minor);
        printf("Name                   : %s\n", devProp.name);
        printf("Total Global Memory    : %d\n", devProp.totalGlobalMem);
        printf("Shared memory per block: %d\n", devProp.sharedMemPerBlock);
        printf("Registers per block    : %d\n", devProp.regsPerBlock);
        printf("Warp size              : %d\n", devProp.warpSize);
        printf("Max threads per block  : %d\n", devProp.maxThreadsPerBlock);
        printf("Total constant memory  : %d\n", devProp.totalConstMem);
        printf("Total mmultiProcessor Count  : %d\n", devProp.multiProcessorCount);
    };

    return;
}

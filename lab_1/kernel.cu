#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <ostream>

void print_info(FILE *);

__global__ void add(int* a, int* b, int* c) 
{
    *c = *a + *b;
}

int main() 
{
    FILE* file = fopen("output.txt", "w");

    int a, b, c;
    // host copies of variables a, b & c
    int* d_a, * d_b, * d_c;
    // device copies of variables a, b & c
    int size = sizeof(int);
    // Allocate space for device copies of a, b, c
    hipMalloc((void**)&d_a, size);
    hipMalloc((void**)&d_b, size);
    hipMalloc((void**)&d_c, size);
    // Setup input values  
    c = int();
    a = 777u;
    b = 333u;
    // Copy inputs to device
    hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);
    // Launch add() kernel on GPU
    add << <1, 1 >> > (d_a, d_b, d_c);
    // Copy result back to host
    hipError_t err = hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);
    if (err != hipSuccess) 
    {
        fprintf(file,"CUDA error copying to Host: %s\n", hipGetErrorString(err));
    }
    fprintf(file,"result sum of %d and %d is %d\n",a , b, c);
    print_info(file);
    fclose(file);
    // Cleanup
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    return 0;
}

void print_info(FILE * f)
{
    int            deviceCount;
    hipDeviceProp_t devProp;

    hipGetDeviceCount(&deviceCount);

    fprintf(f,"Found %d devices\n", deviceCount);

    for (int device = 0; device < deviceCount; device++)
    {
        hipGetDeviceProperties(&devProp, device);

        fprintf(f,"Device %d\n", device);
        fprintf(f,"Compute capability     : %d.%d\n", devProp.major, devProp.minor);
        fprintf(f,"Name                   : %s\n", devProp.name);
        fprintf(f,"Total Global Memory    : %llu\n", devProp.totalGlobalMem);
        fprintf(f,"Shared memory per block: %d\n", devProp.sharedMemPerBlock);
        fprintf(f,"Registers per block    : %d\n", devProp.regsPerBlock);
        fprintf(f,"Warp size              : %d\n", devProp.warpSize);
        fprintf(f,"Max threads per block  : %d\n", devProp.maxThreadsPerBlock);
        fprintf(f,"Total constant memory  : %d\n", devProp.totalConstMem);
        fprintf(f,"Total mmultiProcessor Count  : %d\n", devProp.multiProcessorCount);
    };

    return;
}
